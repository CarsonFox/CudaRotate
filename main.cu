#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>

#include "Image.hpp"

__global__ void rotateNaive(Pixel *in, Pixel *out) {
    const auto i = blockIdx.x * 32 + threadIdx.x;
    const auto j = blockIdx.y * 32 + threadIdx.y;
    out[j * 32 + i] = in[i * 32 + j];
}

void checkErrors(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << hipGetErrorName(err) << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

int main() {
    //Read image from stdin
    Image hostImageIn, hostImageOut;
    std::cin >> hostImageIn;

    //Allocate space for input and output images on device
    Image *devImageIn, *devImageOut;
    checkErrors(hipMalloc(&devImageIn, sizeof(Image)));
    checkErrors(hipMalloc(&devImageOut, sizeof(Image)));

    //Send input image to device
    checkErrors(hipMemcpy(devImageIn, &hostImageIn, sizeof(Image), hipMemcpyHostToDevice));

    dim3 blockDim(32, 32, 1);
    dim3 gridDim(32, 32, 1);

    //Call kernel to rotate image
    rotateNaive<<<gridDim, blockDim>>>((Pixel *)devImageIn, (Pixel *)devImageOut);
    checkErrors(hipPeekAtLastError());

    //Send rotated image back to host
    checkErrors(hipMemcpy(&hostImageOut, devImageOut, sizeof(Image), hipMemcpyDeviceToHost));

    checkErrors(hipDeviceSynchronize());

    //Check that the image was rotated
    assert(isRotated(hostImageIn, hostImageOut));

    //Write rotated image to stdout
//    std::cout << hostImageOut;

    //Cleanup
    hipFree(devImageIn);
    hipFree(devImageOut);

    return 0;
}
