#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>

#include "Image.hpp"

void naive();
void tiled();

__global__ void rotateNaive(Pixel *in, Pixel *out) {
    out[blockIdx.x * IMAGE_SIZE + threadIdx.x] = in[threadIdx.x * IMAGE_SIZE + blockIdx.x];
}

void checkErrors(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << hipGetErrorName(err) << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

int main() {
    naive();
}

void tiled() {
}

void naive() {
    //Read image from stdin
    Image hostImageIn, hostImageOut;
    std::cin >> hostImageIn;

    //Allocate space for input and output images on device
    Image *devImageIn, *devImageOut;
    checkErrors(hipMalloc(&devImageIn, sizeof(Image)));
    checkErrors(hipMalloc(&devImageOut, sizeof(Image)));

    //Send input image to device
    checkErrors(hipMemcpy(devImageIn, &hostImageIn, sizeof(Image), hipMemcpyHostToDevice));

    //Call kernel to rotate image
    rotateNaive<<<IMAGE_SIZE, IMAGE_SIZE>>>((Pixel *)devImageIn, (Pixel *)devImageOut);
    checkErrors(hipPeekAtLastError());

    //Send rotated image back to host
    checkErrors(hipMemcpy(&hostImageOut, devImageOut, sizeof(Image), hipMemcpyDeviceToHost));

    checkErrors(hipDeviceSynchronize());

    //Check that the image was rotated
    assert(isRotated(hostImageIn, hostImageOut));

    //Write rotated image to stdout
//    std::cout << hostImageOut;

    //Cleanup
    hipFree(devImageIn);
    hipFree(devImageOut);
}
