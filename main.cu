#include "hip/hip_runtime.h"
#include <iostream>

#include "Image.hpp"

__global__ void echo() {
    printf("x: %d\ty: %d\n", threadIdx.x, threadIdx.y);
}

void checkErrors(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << hipGetErrorName(err) << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

int main() {
    auto inputImage = new Image(), outputImage = new Image();
    Image *d_image;
    checkErrors(hipMalloc(&d_image, sizeof(Image)));

    std::cin >> *inputImage;

    checkErrors(hipMemcpy(d_image, inputImage, sizeof(Image), hipMemcpyHostToDevice));
    echo<<<4,4>>>();
    checkErrors(hipMemcpy(outputImage, d_image, sizeof(Image), hipMemcpyDeviceToHost));

    checkErrors(hipDeviceSynchronize());

    std::cout << *outputImage;

    delete inputImage;
    delete outputImage;
    hipFree(d_image);
    return 0;
}
