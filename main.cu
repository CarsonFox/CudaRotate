#include "hip/hip_runtime.h"
#include <iostream>

#include "Image.hpp"

__global__ void add(int *a, int *b, int *out) {
    *out = *a + *b;
}

void cudaHello() {
    int a = 0, b = 0, out = 0;
    int *d_a, *d_b, *d_out;

    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));
    hipMalloc(&d_out, sizeof(int));

    a = 5, b = 11;

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    add<<<1,1>>>(d_a, d_b, d_out);

    hipMemcpy(&out, d_out, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a), hipFree(d_b), hipFree(d_out);

    std::cout << a << " + " << b << " = " << out << std::endl;
}

int main() {
    auto inputImage = new Image(), outputImage = new Image();

    std::cin >> *inputImage;

    for (int i = 0; i < IMAGE_SIZE; i++) {
        for (int j = 0; j < IMAGE_SIZE; j++) {
            (*outputImage)[i][j] = (*inputImage)[j][i];
        }
    }

    std::cout << *outputImage;

    return 0;
}
