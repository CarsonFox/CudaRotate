#include <iostream>

#include "Image.hpp"

int main() {
    auto inputImage = new Image(), outputImage = new Image();
    Image *d_image;
    hipMalloc(&d_image, sizeof(Image));

    std::cin >> *inputImage;

    hipMemcpy(d_image, inputImage, sizeof(Image), hipMemcpyHostToDevice);

    //Kernel

    hipMemcpy(outputImage, d_image, sizeof(Image), hipMemcpyDeviceToHost);

    std::cout << *outputImage;

    delete inputImage;
    delete outputImage;
    hipFree(d_image);
    return 0;
}
