#include "hip/hip_runtime.h"
#include <iostream>

#include "Image.hpp"

__global__ void rotateNaive(Pixel *in, Pixel *out) {
    for (int i = 0; i < IMAGE_SIZE; i++) {
        out[i * IMAGE_SIZE + threadIdx.x] = in[threadIdx.x * IMAGE_SIZE + i];
    }
}

void checkErrors(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << hipGetErrorName(err) << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

int main() {
    auto hostImage = new Image();
    std::cin >> *hostImage;

    Image *devImageIn, *devImageOut;
    checkErrors(hipMalloc(&devImageIn, sizeof(Image)));
    checkErrors(hipMalloc(&devImageOut, sizeof(Image)));

    checkErrors(hipMemcpy(devImageIn, hostImage, sizeof(Image), hipMemcpyHostToDevice));

    rotateNaive<<<1, IMAGE_SIZE>>>((Pixel *)devImageIn, (Pixel *)devImageOut);
    checkErrors(hipPeekAtLastError());

    checkErrors(hipMemcpy(hostImage, devImageIn, sizeof(Image), hipMemcpyDeviceToHost));

    checkErrors(hipDeviceSynchronize());

    std::cout << *hostImage;

    delete hostImage;
    hipFree(devImageIn);

    return 0;
}
