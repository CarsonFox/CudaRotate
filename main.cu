#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <algorithm>
#include <vector>

#include "Image.hpp"

void naive();

void tiled();

//Size of a tile in pixels (square root of image size)
constexpr int tileSize = 32;

int main() {
    tiled();
    //naive();
}

__global__ void rotateNaive(Pixel *in, Pixel *out) {
    out[blockIdx.x * IMAGE_SIZE + threadIdx.x] = in[threadIdx.x * IMAGE_SIZE + blockIdx.x];
}

void checkErrors(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << hipGetErrorName(err) << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void tiled() {
    //Read image from stdin
    Image hostImageIn, hostImageOut;
    std::cin >> hostImageIn;

    //Allocate space for input and output images on device
    Image *devImageIn, *devImageOut;
    checkErrors(hipMalloc(&devImageIn, sizeof(Image)));
    checkErrors(hipMalloc(&devImageOut, sizeof(Image)));

    //Allocate tiles for each block
    const int numBlocks = IMAGE_SIZE / tileSize;
    std::vector<Pixel *> tiles(numBlocks, nullptr);
    for (Pixel *tile : tiles)
        checkErrors(hipMalloc(&tile, tileSize * tileSize));

    //Dimensions of grid and tiles
    const dim3 dim(tileSize, tileSize, 1);

    //Send input image to device
    checkErrors(hipMemcpy(devImageIn, &hostImageIn, sizeof(Image), hipMemcpyHostToDevice));

    //Call kernel to rotate image
    rotateNaive<<<IMAGE_SIZE, IMAGE_SIZE>>>((Pixel *) devImageIn, (Pixel *) devImageOut);
    checkErrors(hipPeekAtLastError());

    //Send rotated image back to host
    checkErrors(hipMemcpy(&hostImageOut, devImageOut, sizeof(Image), hipMemcpyDeviceToHost));

    checkErrors(hipDeviceSynchronize());

    //Check that the image was rotated
    assert(isRotated(hostImageIn, hostImageOut));

    //Write rotated image to stdout
    //std::cout << hostImageOut;

    //Cleanup
    hipFree(devImageIn);
    hipFree(devImageOut);
    for (Pixel *tile : tiles)
        hipFree(tile);
}

void naive() {
    //Read image from stdin
    Image hostImageIn, hostImageOut;
    std::cin >> hostImageIn;

    //Allocate space for input and output images on device
    Image *devImageIn, *devImageOut;
    checkErrors(hipMalloc(&devImageIn, sizeof(Image)));
    checkErrors(hipMalloc(&devImageOut, sizeof(Image)));

    //Send input image to device
    checkErrors(hipMemcpy(devImageIn, &hostImageIn, sizeof(Image), hipMemcpyHostToDevice));

    //Call kernel to rotate image
    rotateNaive<<<IMAGE_SIZE, IMAGE_SIZE>>>((Pixel *) devImageIn, (Pixel *) devImageOut);
    checkErrors(hipPeekAtLastError());

    //Send rotated image back to host
    checkErrors(hipMemcpy(&hostImageOut, devImageOut, sizeof(Image), hipMemcpyDeviceToHost));

    checkErrors(hipDeviceSynchronize());

    //Check that the image was rotated
    assert(isRotated(hostImageIn, hostImageOut));

    //Write rotated image to stdout
    //std::cout << hostImageOut;

    //Cleanup
    hipFree(devImageIn);
    hipFree(devImageOut);
}
