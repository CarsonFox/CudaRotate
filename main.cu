#include "hip/hip_runtime.h"
#include <iostream>

#include "Image.hpp"

__global__ void add(int *a, int *b, int *out) {
    *out = *a + *b;
}

void cudaHello() {
    int a = 0, b = 0, out = 0;
    int *d_a, *d_b, *d_out;

    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));
    hipMalloc(&d_out, sizeof(int));

    a = 5, b = 11;

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    add<<<1,1>>>(d_a, d_b, d_out);

    hipMemcpy(&out, d_out, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a), hipFree(d_b), hipFree(d_out);

    std::cout << a << " + " << b << " = " << out << std::endl;
}

int main() {
    cudaHello();

    return 0;
}
