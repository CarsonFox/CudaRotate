#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <algorithm>
#include <vector>

#include "Image.hpp"

void naive();
void tiled();
void checkErrors(hipError_t err);

//Size of a tile in pixels (square root of image size)
constexpr int tileSize = 32;

int main() {
    tiled();
    //naive();
}

__global__ void rotateNaive(Pixel *in, Pixel *out) {
    out[blockIdx.x * IMAGE_SIZE + threadIdx.x] = in[threadIdx.x * IMAGE_SIZE + blockIdx.x];
}

__global__ void writeToTiles(Pixel *in, Pixel *tiles) {
     auto in_i = blockIdx.x * blockDim.x + threadIdx.x;
     auto in_j = blockIdx.y * blockDim.y + threadIdx.y;
     auto in_index = in_i + in_j * IMAGE_SIZE;

     auto tile_i = blockIdx.x * blockDim.x + threadIdx.y;
     auto tile_j = blockIdx.y * blockDim.y + threadIdx.x;
     auto tile_index = tile_i + tile_j * IMAGE_SIZE;

     tiles[tile_index] = in[in_index];
}

__global__ void readFromTiles(Pixel *tiles, Pixel *out) {
    auto tile_i = blockIdx.x * blockDim.x + threadIdx.x;
    auto tile_j = blockIdx.y * blockDim.y + threadIdx.y;
    auto tile_index = tile_i + tile_j * IMAGE_SIZE;

    auto out_i = blockIdx.y * blockDim.x + threadIdx.x;
    auto out_j = blockIdx.x * blockDim.y + threadIdx.y;
    auto out_index = out_i + out_j * IMAGE_SIZE;

    out[out_index] = tiles[tile_index];
}

void tiled() {
    //Read image from stdin
    Image hostImageIn, hostImageOut;
    std::cin >> hostImageIn;

    //Allocate space for input and output images on device
    Image *devImageIn, *devImageOut;
    checkErrors(hipMalloc(&devImageIn, sizeof(Image)));
    checkErrors(hipMalloc(&devImageOut, sizeof(Image)));

    //Allocate tiles for each block.
    //Although memory will be allocated contiguously, treating it like
    //an array of tiles will result in smaller strides.
    Pixel *tiles;
    checkErrors(hipMalloc(&tiles, sizeof(Image)));

    //Dimensions of grid and tiles
    const dim3 dim(tileSize, tileSize, 1);

    //Send input image to device
    checkErrors(hipMemcpy(devImageIn, &hostImageIn, sizeof(Image), hipMemcpyHostToDevice));

    //Call kernels to rotate image
    writeToTiles<<<dim, dim>>>((Pixel *) devImageIn, tiles);
    readFromTiles<<<dim, dim>>>(tiles, (Pixel *) devImageOut);
    checkErrors(hipPeekAtLastError());

    //Send rotated image back to host
    checkErrors(hipMemcpy(&hostImageOut, devImageOut, sizeof(Image), hipMemcpyDeviceToHost));

    checkErrors(hipDeviceSynchronize());

    //Check that the image was rotated
    assert(isRotated(hostImageIn, hostImageOut));

    //Write rotated image to stdout
    //std::cout << hostImageOut;

    //Cleanup
    hipFree(devImageIn);
    hipFree(devImageOut);
    hipFree(tiles);
}

void naive() {
    //Read image from stdin
    Image hostImageIn, hostImageOut;
    std::cin >> hostImageIn;

    //Allocate space for input and output images on device
    Image *devImageIn, *devImageOut;
    checkErrors(hipMalloc(&devImageIn, sizeof(Image)));
    checkErrors(hipMalloc(&devImageOut, sizeof(Image)));

    //Send input image to device
    checkErrors(hipMemcpy(devImageIn, &hostImageIn, sizeof(Image), hipMemcpyHostToDevice));

    //Call kernel to rotate image
    rotateNaive<<<IMAGE_SIZE, IMAGE_SIZE>>>((Pixel *) devImageIn, (Pixel *) devImageOut);
    checkErrors(hipPeekAtLastError());

    //Send rotated image back to host
    checkErrors(hipMemcpy(&hostImageOut, devImageOut, sizeof(Image), hipMemcpyDeviceToHost));

    checkErrors(hipDeviceSynchronize());

    //Check that the image was rotated
    assert(isRotated(hostImageIn, hostImageOut));

    //Write rotated image to stdout
    //std::cout << hostImageOut;

    //Cleanup
    hipFree(devImageIn);
    hipFree(devImageOut);
}

void checkErrors(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << hipGetErrorName(err) << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
